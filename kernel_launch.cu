#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 500000
__global__ void shortKernel(float *out_d, float *in_d, int num)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num)
    {
        out_d[idx] = 1.23 * in_d[idx];
        // out_d[idx] = 1.0f;
    }
}

#define NSTEP 10
#define NKERNEL 20


void graph_launch()
{
    hipStream_t stream;
    hipStreamCreate(&stream);

    bool graphCreated = false;
    hipGraph_t cuGraph;
    hipGraphExec_t cuGraphInstance;

    int size = N * sizeof(float);
    hipError_t cuError;
    float *in_d;
    float *out_d;

    float *in_h = (float*)malloc(size);
    float *out_h = (float*)malloc(size);

    for (int i = 0; i < N; i++)
    {
        in_h[i] = rand() / (float)RAND_MAX;
    }

    cuError = hipMalloc((void**)&in_d, size);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    cuError = hipMalloc((void**)&out_d, size);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    cuError = hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    for (int istep = 0; istep < NSTEP; istep++)
    {
        if (!graphCreated)
        {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for (int ikrnl = 0; ikrnl < NKERNEL; ikrnl++)
            {
                shortKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(out_d, in_d, N);
            }
            hipStreamEndCapture(stream, &cuGraph);
            hipGraphInstantiate(&cuGraphInstance, cuGraph, NULL, NULL, 0);
            graphCreated = true;
        }
        hipGraphLaunch(cuGraphInstance, stream);
        hipStreamSynchronize(stream);
    }
}

int main()
{
    graph_launch();
    return 0;
    int size = N * sizeof(float);
    hipError_t cuError;
    float *in_d;
    float *out_d;

    float *in_h = (float*)malloc(size);
    float *out_h = (float*)malloc(size);

    for (int i = 0; i < N; i++)
    {
        in_h[i] = rand() / (float)RAND_MAX;
    }

    cuError = hipMalloc((void**)&in_d, size);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    cuError = hipMalloc((void**)&out_d, size);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    cuError = hipMemcpy(in_d, in_h, size, hipMemcpyHostToDevice);
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to malloc (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    // cudaStream_t streams[14];
    // for (int i = 0; i < 14; i++)
    // {
    //     cudaStreamCreate(&streams[i]);
    // }
    

    // int blocks = 256;
    // int threads = 16;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    for (int istep = 0; istep < NSTEP; istep++)
    {
        for (int ikrnl = 0; ikrnl < NKERNEL; ikrnl++)
        {
            shortKernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(out_d, in_d, N);
            // cudaStreamSynchronize(stream);
        }
        hipStreamSynchronize(stream);
    }

    cuError = hipGetLastError();
    if (cuError != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(cuError));
        exit(EXIT_FAILURE);
    }

    hipMemcpy(out_h, out_d, N * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < N; i++)
    // {
    //     printf("%f ", out_h[i]);
    // }
    // printf("\n");
    
    hipFree(in_d);
    hipFree(out_d);
    free(in_h);
    free(out_h);

    return 0;
}